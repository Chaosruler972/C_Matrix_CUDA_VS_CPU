#include "hip/hip_runtime.h"

/*
	CUDA libraries
*/
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "matrix.h" // the class representing matrix along with the multipications
#define CUDA_BLOCK 16 // cuda GPU block size, for CUDA multithreadding

#define RATIO_SECOND_TO_MS 0.001 // to convert ms (from cuda events) to seconds (human vision)



int main()
{
	int mat_size = MAX_MAT_SIZE;

	/*
		matrixes to multiply
		their size is N*N which will be MAX_MAT_SIZE from matrix.h
	*/
	matrix m1(mat_size);
	matrix m2(mat_size);

	hipEvent_t device_begin, device_end; // calculate function processing time

	/*
		create event recorders -- should call Destroyer...
	*/
	hipEventCreate(&device_begin); 
	hipEventCreate(&device_end);
	clock_t start;
	clock_t end;
	/*
		Satistics variables
	*/
	float GPU_elapsed = 0, CPU_multithreadded_elapsed = 0;// CPU_single_elapsed = 0;

	int i, j; // iterators, used for random variables looping through both matrixes
	/*
		init mat1 and mat2 stage
	*/
	for (i = 0; i < mat_size; i++) // init matrix
	{
		/*
			random number is between 0 and RAND_MAX at matrix.h
		*/
		for (j = 0; j < mat_size; j++)
		{
			m1[i][j] = random_number(); // random number for matrix m1 at index i,j; 
			m2[i][j] = random_number(); // random number for matrix m2 at index i,j;
		}
	}


	/*
		CPU single core mult
	*/
	/*
	start = clock(); // parallel CPU clock counting, just in case // switched to main
	hipEventRecord(device_begin, 0); // cpu single start recording
	matrix m3_cpu_single = m1 * m2; // cpu single core mult
	hipEventRecord(device_end, 0); // cpu single stop recording
	hipEventSynchronize(device_end); // sync
	end = clock(); // parallel CPU clock counting, just in case // switched to main
	CPU_single_elapsed = (end - start) / CLOCKS_PER_SEC;
	if (CPU_single_elapsed == 0)
	{
		hipEventElapsedTime(&CPU_single_elapsed, device_begin, device_end); // calculate
		CPU_single_elapsed *= RATIO_SECOND_TO_MS; // convert to second from ms
	}
	//satistics
	std::cout << " Host (CPU single threadded) calculated matrix multipication at size of " << mat_size << "X" << mat_size << " at " << CPU_single_elapsed << " Seconds" << std::endl;
	*/

	/*
		CPU multicore mult
	*/
	start = clock(); // parallel CPU clock counting, just in case // switched to main
	hipEventRecord(device_begin, 0); // cpu multi start recording
	matrix m3_cpu_multi = m1 * m2; // cpu multicore mult
	hipEventRecord(device_end, 0); // cpu multi stop recording
	hipEventSynchronize(device_end); // sync
	end = clock(); // parallel CPU clock counting, just in case // switched to main
	CPU_multithreadded_elapsed =(float) ( (end - start) / CLOCKS_PER_SEC);
	if (CPU_multithreadded_elapsed == 0)
	{
		hipEventElapsedTime(&CPU_multithreadded_elapsed, device_begin, device_end); // calculate
		CPU_multithreadded_elapsed *= (float) RATIO_SECOND_TO_MS; // convert to second from ms
	}
	//sattistics
	std::cout << " Host (CPU Multithreadded!) calculated matrix multipication at size of " << mat_size << "X" << mat_size << " at " << CPU_multithreadded_elapsed << " Seconds" << std::endl;

	
	
	/*
				CUDA mult
	*/

	hipEventRecord(device_begin, 0); // start recording
	matrix m3_gpu = m1.cuda_mult(m2); // cuda mult
	hipEventRecord(device_end, 0); // stop recording
	hipEventSynchronize(device_end); // sync
	hipEventElapsedTime(&GPU_elapsed, device_begin, device_end); // calculate
	GPU_elapsed *= (float) RATIO_SECOND_TO_MS; // convert to second from ms
	//satisitcs
	std::cout << " Device (GPU) calculated matrix multipication at size of " << mat_size << "X" << mat_size << " at " << GPU_elapsed << " Seconds" << std::endl;


	hipEventDestroy(device_begin); // destroy recorders
	hipEventDestroy(device_end);
	
	/*
	    matrix printers, for testing
	*/

	/*
	std::cout << "CPU Single: " << m3_cpu_single << std::endl << std::endl;
	std::cout << "CPU Multi: " << m3_cpu_multi << std::endl << std::endl;
	std::cout << "GPU: "       << m3_gpu << std::endl << std::endl;
	*/


	/* 
			satistics printing!
	*/
	std::cout << std::endl;
	std::cout << std::endl;

	/*
			Correctness
	*/
	std::cout << "And both matrixes are ";
	if (m3_cpu_multi == m3_gpu)
		std::cout << " Equal, therefore correct" << std::endl;
	else
		std::cout << " Unequal, therefore (most likely GPU) is incorrect " << std::endl;

	std::cout << std::endl;
	std::cout << std::endl;

	/*
			RATIO - GPU and CPU multi-thread
	*/

	std::cout << " The ratio between GPU and CPU (multi_threadded) (GPU/CPU Multi-threadded) is  ";
	if (CPU_multithreadded_elapsed == 0)
		std::cout << " Uncountable";
	else
		std::cout << (GPU_elapsed) / (CPU_multithreadded_elapsed) << std::endl;


	/*

	/*
				ratio - CPU single vs CPU multi 
	/*

	std::cout << std::endl;
	std::cout << std::endl;

	std::cout << " Also, ratio between CPU multicore and CPU single core (CPU_MULTI/CPU_SINGLE) ";

	if (CPU_single_elapsed == 0)
		std::cout << " Uncountable";
	else
		std::cout << (CPU_multithreadded_elapsed) / (CPU_single_elapsed) << std::endl;

	std::cout << std::endl;
	std::cout << std::endl;

	
			ratio - GPU vs CPU single core
	

	std::cout << " Also, ratio between GPU and CPU single core (GPU/CPU_SINGLE) ";

	if (CPU_single_elapsed == 0)
		std::cout << " Uncountable";
	else
		std::cout << (GPU_elapsed) / (CPU_single_elapsed) << std::endl;
	*/
	return 0;
}



__global__ void cuda_matrix_mult(int* m1, int* m2, int* m3, int size)
{
	int row = threadIdx.x;
	int col = blockIdx.x;
	if (row >= size || col >= size)
		return;
	for (int i = 0; i < size; i++)
	{
		m3[row*size + col] += m1[row*size + i] * m2[i*size + col];
	}
}


matrix matrix::cuda_mult(const matrix &other)
{
	matrix matrix3(this->N); // store results here
	if (this->N != other.N)
		return matrix3;
	int *device_a=NULL, *device_b=NULL, *device_c=NULL;
	int **host_a = NULL, **host_b = NULL, **host_c = NULL;
	int size = this->N;
	int i, j;
	/*
		allocate main pointers
	*/
	hipHostMalloc((void **)&host_a, sizeof(int*)*size); // cpu-gpu memory bridge
	hipHostMalloc((void **)&host_b, sizeof(int*)*size);
	hipHostMalloc((void **)&host_c, sizeof(int*)*size);

	for (i = 0; i < size; i++)
	{
		/*
			2d array - allocate inner arrays
		*/
		hipHostMalloc((void **)&host_a[i], sizeof(int)*size);
		hipHostMalloc((void **)&host_b[i], sizeof(int)*size);
		hipHostMalloc((void **)&host_c[i], sizeof(int)*size);
	}


	/*
		copy data from this and other to host device pointer bridges
	*/
	for (i = 0; i < size; i++)
	{
		for (j = 0; j < size; j++)
		{
			host_a[i][j] = this->arr[i][j];
			host_b[i][j] = other.arr[i][j];
			host_c[i][j] = matrix3.arr[i][j];
		}
	}




	hipMalloc((void**) &device_a, sizeof(int)*size*size); // gpu memory allloc
	hipMalloc((void**) &device_b, sizeof(int)*size*size);
	hipMalloc((void**) &device_c, sizeof(int)*size*size);


	

	for (int i = 0; i < size; i++)
	{
		
		hipMemcpy(device_a+i*size, host_a[i], sizeof(int)*size, hipMemcpyHostToDevice); // copy mem to gpu
		hipMemcpy(device_b+i*size, host_b[i], sizeof(int)*size, hipMemcpyHostToDevice);
		hipMemcpy(device_c+i*size, host_c[i], sizeof(int)*size, hipMemcpyHostToDevice);
	}

		
	
	/* multi threadded mult call
	*/
	cuda_matrix_mult <<<size,size >>>(device_a,device_b,device_c,size); // function call
	
	hipDeviceSynchronize();

	for (i = 0; i < size; i++)
	{
		hipMemcpy(host_c[i], device_c+i*size, sizeof(int)*size, hipMemcpyDeviceToHost); // copy results back

	}

	/*
		copy it to class
	*/
	for (i = 0; i < size; i++)
	{
		for (j = 0; j < size; j++)
		{
			matrix3[i][j] = host_c[i][j];
		}
	}

	hipDeviceSynchronize();
	for (i = 0; i < size; i++)
	{
		/*
			free inner
		
		hipFree(device_a[i]);
		hipFree(device_b[i]);
		hipFree(device_c[i]);
		*/
		hipHostFree(host_a[i]);
		hipHostFree(host_b[i]);
		hipHostFree(host_c[i]);
	}
	/*
		free-outer
	*/
	hipFree(device_a);
	hipFree(device_b);
	hipFree(device_c);
	hipHostFree(host_a);
	hipHostFree(host_b);
	hipHostFree(host_c);

	return matrix3;
}
